#include "hip/hip_runtime.h"
#include "Physics.cuh"

#include <vector>

class NBody_cuda 
{
    private:
        int _N;
        float _tf;
        int _timeSteps;
        float _deltaT;
    public:
        std::vector<Body*> _bodies;
        
        NBody(int, float, int);
        ~NBody() 
        { 
            for(int i=0; _bodies.size(); i++)
            {
                delete _bodies[i];
            }
        }

        void print_cycle(float);
        void print_sim();

        //Physics
        void display_bodies();
        void compute();
        __device__
        void d_updateAcceleration(int, Position3D *,Acceleration3D *,Mass *);

        __device__
        void d_updateVelocity(int , float , Acceleration3D *, Velocity3D *); 


        __device__
        void d_updatePosition(int, float, Velocity3D *,Position3D *); 


        __global__
        void d_updatePhysics(int, float, Position3D *, Velocity3D *,Acceleration3D *,Mass *);

        //getters and setters
        int get_N() const {return _N;}
        int getTimeSteps() const {return _timeSteps;}; 
};
    