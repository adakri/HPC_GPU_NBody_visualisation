#include "hip/hip_runtime.h"
#include "Physics.cuh"

#include <vector>
#include <chrono>

//https://gist.github.com/lebedov/bca3c70e664f54cdf8c3cd0c28c11a0f

class NBody_cuda 
{
    private:
        int _N;
        float _tf;
        int _timeSteps;
        float _deltaT;
    public:
        std::vector<Body*> _bodies;
        
        NBody_cuda(int, float, int);
        ~NBody_cuda() 
        { 
            for(int i=0; _bodies.size(); i++)
            {
                delete _bodies[i];
            }
        };

        void print_cycle(float);
        void print_sim();

        //Physics
        void display_bodies();
        void setUP_cuda();

        //getters and setters
        int get_N() const {return _N;}
        int getTimeSteps() const {return _timeSteps;}; 
};


// I did not know which extension to use .cpp or .cu ??
// déclarons les dans le meme fichier


//constructor
NBody_cuda::NBody_cuda(int Nu , float tf, int timeSteps): _N(Nu), _tf(tf), _timeSteps(timeSteps) 
{
};

void NBody_cuda::display_bodies()
{
  for( int i = 0; i < _N; i++ ) 
  {
    printf("\nBody %d:\nMass: %f\nPosition(x ,y, z): %f, %f, %f\nVelocity(x, y, z): %f, %f, %f\nAcceleration(x ,y, z): %f, %f, %f\n\n",
      i + 1, 
      _bodies[i]->_mass,
      _bodies[i]->_position._x, _bodies[i]->_position._y, _bodies[i]->_position._z,

      _bodies[i]->_velocity._x, _bodies[i]->_velocity._y, _bodies[i]->_velocity._z,
      _bodies[i]->_acceleration._x, _bodies[i]->_acceleration._y, _bodies[i]->_acceleration._z);
  }
};


// compute acceleration per thread
__device__
void d_updateAcceleration(int index, vector *d_pos,vector *d_acc, Mass *d_mass, int _N) 
{
   
  vector netForce = { 0, 0, 0 };

  for( int i = 0; i < _N; i++ ) 
  {
    if( i == index ) 
    {
      continue;
    }

    vector vectorForceToOther = {0, 0, 0};

    Force scalarForceBetween = ComputeForce(
                                  d_mass[index],
                                  d_mass[i],
                                  d_pos[index],
                                  d_pos[i]);



    v_direction( d_pos[index],d_pos[i],vectorForceToOther);

    vectorForceToOther._x *= scalarForceBetween;
    vectorForceToOther._y *= scalarForceBetween;
    vectorForceToOther._z *= scalarForceBetween;

    netForce._x += vectorForceToOther._x;
    netForce._y += vectorForceToOther._y;
    netForce._z += vectorForceToOther._z;
  }

  d_acc[index] = computeAcceleration3D(d_mass[index], netForce);
};

__device__
void d_updateVelocity(int index, float deltaT, vector *d_acc, vector *d_vel) 
{
  d_vel[index] = computeVelocity3D(
                                d_acc[index],
                                d_vel[index],
                                deltaT);
};

__device__
void d_updatePosition(int index, float deltaT, vector *d_vel, vector *d_pos) 
{

  d_pos[index] = computePosition3D( 
                              d_vel[index],
                              d_pos[index],
                              deltaT);
};


__global__
void updatePhysics(int bodies, float deltaT, vector *d_pos, vector *d_vel, vector *d_acc, Mass *d_mass, int _N)
{
  
  // 1D blocks and thread organisation
  int blockidx = blockIdx.x;
  int threadidx = threadIdx.x;

  int element_id = (blockidx * threadidx) + threadidx;

  if(element_id > _N)
    return;
  
  d_updateAcceleration(element_id, d_pos, d_acc, d_mass, _N);
  d_updateVelocity(element_id, deltaT, d_acc, d_vel);
  d_updatePosition(element_id, deltaT, d_vel, d_pos);
};


// The execution is very different 
void NBody_cuda::setUP_cuda()
{  

  // the C syntax
  int VECTOR_SIZE_IN_BYTES = _N * sizeof(vector);
  int SCALAR_SIZE_IN_BYTES = _N * sizeof(Scalar);

  //Initializing Velocities of N bodies in GPU
  vector *h_vel = nBodyVelocity;
  vector *d_vel;
  hipMalloc((void**) &d_vel, VECTOR_SIZE_IN_BYTES);
  hipMemcpy(d_vel, h_vel, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice);

  //Initializing acceleration of N bodies in GPU
  vector *h_acc = nBodyAcceleration;
  vector *d_acc;
  hipMalloc((void**) &d_acc, VECTOR_SIZE_IN_BYTES);
  hipMemcpy(d_acc, h_acc, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice);
  
  //Initializing Mass of N bodies in GPU
  Mass *h_mass = nBodyMass;
  Mass *d_mass;
  hipMalloc((void**) &d_mass, SCALAR_SIZE_IN_BYTES);
  hipMemcpy(d_mass, h_mass, SCALAR_SIZE_IN_BYTES, hipMemcpyHostToDevice);

    //Initializing Positions of N bodies in GPU
  vector *h_pos = nBodyPosition;
  vector *d_pos;
  hipMalloc((void**) &d_pos, VECTOR_SIZE_IN_BYTES);
  hipMemcpy(d_pos, h_pos, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice);



  for (int i = 0; i < 10000; ++i)
  {
    updatePhysics<<<(_N/16) + 1, 16>>>(_N, (float)(i * 100), d_pos, d_vel, d_acc, d_mass, _N);
  }
    

  hipMemcpy(h_pos, d_pos, VECTOR_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_vel, d_vel, VECTOR_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_acc, d_acc, VECTOR_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
  hipFree(d_pos);
  hipFree(d_vel);
  hipFree(d_acc);
  hipFree(d_mass);

}

int main() 
{
  NBody_cuda nbody = NBody_cuda(N,1.,2);

  std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
  
  nbody.setUP_cuda();
  
  std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

  //Computing the time
  std::cout << "Time taken by the CUDA kernel is " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;
}