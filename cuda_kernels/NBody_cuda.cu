#include "hip/hip_runtime.h"
#include "Physics.cuh"
#include "GL_context.hpp"

#include <vector>
#include <chrono>
#include<unistd.h>



bool DEMO_CUDA = true;




//https://gist.github.com/lebedov/bca3c70e664f54cdf8c3cd0c28c11a0f

class NBody_cuda 
{
    private:
        int _N;
        float _tf;
        int _timeSteps;
        float _deltaT;
    public:
        std::vector<Body*> _bodies;
        
        NBody_cuda(int, float, int);
        ~NBody_cuda() 
        { 
            for(int i=0; _bodies.size(); i++)
            {
                delete _bodies[i];
            }
        };

        void print_cycle(float);
        void print_sim();

        //Physics
        void display_bodies();
        void setUP_cuda(float , vector *, vector *, vector *, Mass *);

        //getters and setters
        int get_N() const {return _N;}
        int getTimeSteps() const {return _timeSteps;}; 
};


// I did not know which extension to use .cpp or .cu ??
// déclarons les dans le meme fichier


//constructor
NBody_cuda::NBody_cuda(int N_b , float tf, int timeSteps): _N(N_b), _tf(tf), _timeSteps(timeSteps) 
{
  _bodies.resize(_N);
  if(!DEMO_CUDA)
  {
    for(int i=1; i<_N; i++)
    {
      _bodies[i] = new Body(
        Vec3(nBodyPosition[i]._x,nBodyPosition[i]._y,nBodyPosition[i]._z),randomParticleVelocity(),
        Vec3(nBodyVelocity[i]._x, nBodyVelocity[i]._y, nBodyVelocity[i]._z),
        nBodyMass[i], random_radius()
                                    );
    }
  }else{
    printf("Initializing begun \n");
    _bodies[0] = new Body(Vec3(0.,0.,0.),
                                 Vec3(0.,0.,0.),
                                  randomParticleacceleration(),
                                  10e16,
                                  35
                                  );
    for(int i=1; i<_N; i++)
    {
      _bodies[i] = new Body(randomParticlePosition(),
                            randomParticleVelocity(),
                              randomParticleacceleration(),
                              random_mass(),
                              random_radius()
                              );
    }
  }
  
  std::cout<<"=====Constructed the NBody class in cuda===="<<std::endl;
};

// useless tbh
void NBody_cuda::display_bodies()
{

  for( int i = 0; i < _N; i++ ) 
  {
    printf("\nBody %d:\nMass: %f\nPosition(x ,y, z): %f, %f, %f\nVelocity(x, y, z): %f, %f, %f\nAcceleration(x ,y, z): %f, %f, %f\n\n",
      i + 1, 
      _bodies[i]->_mass,
      _bodies[i]->_position._x, _bodies[i]->_position._y, _bodies[i]->_position._z,

      _bodies[i]->_velocity._x, _bodies[i]->_velocity._y, _bodies[i]->_velocity._z,
      _bodies[i]->_acceleration._x, _bodies[i]->_acceleration._y, _bodies[i]->_acceleration._z);
  }
};


// compute acceleration per thread
__device__
void d_updateAcceleration(int index, vector *d_pos,vector* d_vel, vector *d_acc, Mass *d_mass, int _N) 
{
   
  vector netForce = { 0, 0, 0 };

  for( int i = 0; i < _N; i++ ) 
  {
    if( i == index ) 
    {
      continue;
    }

    vector vectorForceToOther = {0., 0., 0.};

    Force scalarForceBetween = ComputeForce(
                                  d_mass[index],
                                  d_mass[i],
                                  d_pos[index],
                                  d_pos[i],
                                  d_vel[index]);

    //printf("scalarForceBetween %d : %d %f \n",index, i, scalarForceBetween);

    v_direction( d_pos[index],d_pos[i],vectorForceToOther);

    vectorForceToOther._x *= scalarForceBetween;
    vectorForceToOther._y *= scalarForceBetween;
    vectorForceToOther._z *= scalarForceBetween;

    //printf("The vectorForceToOther %d: %d: %f %f %f \n", index, i, vectorForceToOther._x, vectorForceToOther._y, vectorForceToOther._z);

    netForce._x += vectorForceToOther._x;
    netForce._y += vectorForceToOther._y;
    netForce._z += vectorForceToOther._z;

    //printf("The netForce %d: %d: %f %f %f \n", index, i, netForce._x, netForce._y, netForce._z);
  }

  d_acc[index] = computeAcceleration3D(d_mass[index], netForce);

  //printf("The accelerations: %d : %f %f %f \n", index,d_acc[index]._x, d_acc[index]._y, d_acc[index]._z);
};

__device__
void d_updateVelocity(int index, float deltaT, vector *d_acc, vector *d_vel) 
{
  d_vel[index] = computeVelocity3D(
                                d_acc[index],
                                d_vel[index],
                                deltaT);
  //printf("The velocities: %d : %f %f %f \n", index,d_vel[index]._x, d_vel[index]._y, d_vel[index]._z);
};

__device__
void d_updatePosition(int index, float deltaT, vector *d_vel, vector *d_pos) 
{

  d_pos[index] = computePosition3D( 
                              d_vel[index],
                              d_pos[index],
                              deltaT);
  printf("The positions: %d : %f %f %f \n", index,d_pos[index]._x, d_pos[index]._y, d_pos[index]._z);
};


__global__
void updatePhysics(int bodies, float deltaT, vector *d_pos, vector *d_vel, vector *d_acc, Mass *d_mass, int _N)
{
  printf("Inside the update physics \n");
  // 1D blocks and thread organisation
  int blockidx = blockIdx.x;
  int threadidx = threadIdx.x;

  int element_id = (blockidx * threadidx) + threadidx;

  //printf("the element id %d \n", element_id);

  if(element_id > _N)
    return;

  //printf("%s \n", "===== The positions in the updatePhys b4==== \n");
  /* for(int i=1; i<element_id; i++)
  {
    d_pos++;
  } */
  //printf("@ %d %f, %f, %f \n",element_id,d_pos->_x,d_pos->_y,d_pos->_z);


  d_updateAcceleration(element_id, d_pos, d_vel, d_acc, d_mass, _N);
  d_updateVelocity(element_id, deltaT, d_acc, d_vel);
  d_updatePosition(element_id, deltaT, d_vel, d_pos);

  //printf("%s \n", "===== The positions in the updatePhys==== \n");
  /* printf("* %d %f, %f, %f \n",element_id,(d_pos+element_id)->_x,(d_pos+element_id)->_y,(d_pos+element_id)->_z); */

};


// Determine first time acess
bool FIRST_TIME = true;


// The execution is very different 
// Compute through GPU the update at time time and update the body ppoisitons
void NBody_cuda::setUP_cuda(float time, vector *h_pos, vector *h_vel, vector *h_acc, Mass *h_mass)
{  

  // the C syntax
  int VECTOR_SIZE_IN_BYTES = _N * sizeof(vector);
  int SCALAR_SIZE_IN_BYTES = _N * sizeof(Scalar);

  //Initializing Velocities of N bodies in GPU
  //vector *h_vel = nBodyVelocity;
  vector *d_vel;
  hipMalloc((void**) &d_vel, VECTOR_SIZE_IN_BYTES);
  hipMemcpy(d_vel, h_vel, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice);

  //Initializing acceleration of N bodies in GPU
  //vector *h_acc = nBodyAcceleration;
  vector *d_acc;
  hipMalloc((void**) &d_acc, VECTOR_SIZE_IN_BYTES);
  hipMemcpy(d_acc, h_acc, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice);
  
  //Initializing Mass of N bodies in GPU
  //Mass *h_mass = nBodyMass;
  Mass *d_mass;
  hipMalloc((void**) &d_mass, SCALAR_SIZE_IN_BYTES);
  hipMemcpy(d_mass, h_mass, SCALAR_SIZE_IN_BYTES, hipMemcpyHostToDevice);

  //Initializing Positions of N bodies in GPU
  //vector *h_pos = nBodyPosition;
  vector *d_pos;
  hipMalloc((void**) &d_pos, VECTOR_SIZE_IN_BYTES);
  hipMemcpy(d_pos, h_pos, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice);


  // Print and update
  /*  std::cout<<"===== The starting positions===="<<std::endl;
  for(int i=0; i<_N; i++)
  {
    std::cout<<(h_pos+i)->_x<<","<<(h_pos+i)->_y<<","<<(h_pos+i)->_z<<std::endl;
  } */



  //exit(0);

  printf("The number of 32 thread blocks %d \n", (int)ceil(_N/32));

  std::cout << "AT time "<<time << std::endl;
  
  updatePhysics<<<(int)ceil(_N/32), 32>>>(_N, time*100., d_pos, d_vel, d_acc, d_mass, _N);

    

  hipMemcpy(h_pos, d_pos, VECTOR_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_vel, d_vel, VECTOR_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_acc, d_acc, VECTOR_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
  hipFree(d_pos);
  hipFree(d_vel);
  hipFree(d_acc);
  hipFree(d_mass);

  /* std::cout<<"===== The Final positions===="<<std::endl;
  for(int i=0; i<_N; i++)
  {
    std::cout<<h_pos->_x<<","<<h_pos->_y<<","<<h_pos->_z<<std::endl;
    _bodies[i]->_position = {h_pos->_x, h_pos->_y,h_pos->_z};
    h_pos++; 
  } */
  
  //printf("Finsihed Modifying the positions, thses ones \n");
  for(int i=0; i<_N; i++)
  {
    //std::cout<<(h_pos+i)->_x<<","<<(h_pos+i)->_y<<","<<(h_pos+i)->_z<<std::endl;
    _bodies[i]->_position = {(h_pos+i)->_x, (h_pos+i)->_y,(h_pos+i)->_z};
  }
}


// Class pointer
NBody_cuda* nbody_cuda = new NBody_cuda(Nu,10.,2);


//Initializing Qs of N bodies in GPU
vector *h_vel = nBodyVelocity;
vector *h_acc = nBodyAcceleration;
Mass *h_mass = nBodyMass;
vector *h_pos = nBodyPosition;


// Drawing the bodies through a class pointer is impossibléééé !!

void drawBodies( CStopWatch *timeKeeper, M3DVector4f *lightPosition) {
  // compute displacement and new vectors
  static float previousTime = 0.0f;
  //sleep(3); 
  float currentTime = timeKeeper->GetElapsedSeconds();

  printf("In the set up of the bodies \n");
  

  /* std::cout<<"===== The Present positions===="<<std::endl;
  for(int i=0; i<Nu; i++)
  {
    std::cout<<(h_pos+i)->_x<<","<<(h_pos+i)->_y<<","<<(h_pos+i)->_z<<std::endl;
  } */

  std::cout<<"The time "<<currentTime<<std::endl;

  nbody_cuda->setUP_cuda( currentTime, h_pos, h_vel, h_acc,h_mass );
  
  previousTime = currentTime;

  std::cout<<"#### In the GL Draw ####"<<std::endl;

  for( int i = 0; i < nbody_cuda->get_N() ; i++ ) {
    // Save
    sModelViewMatrixStack.PushMatrix();
    sModelViewMatrixStack.Translate(0.0f, 0.0f, -1000.0f);
    sModelViewMatrixStack.Rotate(1.0f, 0.0f, 1.0f, 1.0f);

    // update position with regard to new values
    sBodyFrames[i].SetOrigin( nbody_cuda->_bodies[i]->_position._x,
                              nbody_cuda->_bodies[i]->_position._y,
                              nbody_cuda->_bodies[i]->_position._z );

    std::cout<<nbody_cuda->_bodies[i]->_position._x<<","<<
            nbody_cuda->_bodies[i]->_position._y<<","<<
            nbody_cuda->_bodies[i]->_position._z<<std::endl; 
    // draw
    sModelViewMatrixStack.MultMatrix( sBodyFrames[i] );
    sShaderManager.UseStockShader( GLT_SHADER_POINT_LIGHT_DIFF,
                                    sTransformPipeline.GetModelViewMatrix(),
                                    sTransformPipeline.GetProjectionMatrix(),
                                    lightPosition,
                                    sBodyColors[0] );
    sBodyBatch[i].Draw();
    // Restore
    sModelViewMatrixStack.PopMatrix();
  }
}

static bool paused = false;

void handleKeypress(unsigned char key, int x, int y) 
{  
  switch (key) 
  {
          case 27:                
            exit(0);              
          case 'p':
            paused = !paused;
            sleep(100);
            printf("Paused for 100s \n");
            
  }
}



void onRenderScene( void ) {
   // Clear the buffer
   glClear( GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT );
   static CStopWatch timeKeeper;
   // Place camera
   M3DMatrix44f mCamera;
   sCameraFrame.GetCameraMatrix( mCamera );
   sModelViewMatrixStack.PushMatrix( mCamera );
   // Transform the light position into eye coordinates
   M3DVector4f lightPos = { sMainLightPos[0],
                            sMainLightPos[1],
                            sMainLightPos[2],
                            sMainLightPos[3]
                          };
   M3DVector4f lightEyePos;
   m3dTransformVector4( lightEyePos, lightPos, mCamera );
   // Call the drawing functions
   drawBodies( &timeKeeper, &lightEyePos );
   // Switch the buffers to bring the drawing on screen
   glutSwapBuffers();
   glutPostRedisplay();
}


void registerCallbacks() {
   glutReshapeFunc( onChangeSize );
   glutDisplayFunc( onRenderScene );
   glutKeyboardFunc(handleKeypress);
}


// Our function
void setupBodies()
{
    std::cout<<"#### In the GL ####"<<std::endl;
    for( int i = 0; i < nbody_cuda->get_N(); i++ ) {
            // Porting the local variable to GPU
            sBodyRadius[i] = nbody_cuda->_bodies[i]->_radius;

            gltMakeSphere( sBodyBatch[i], sBodyRadius[i], 30, 50 );
            sBodyFrames[i].SetOrigin( 
            nbody_cuda->_bodies[i]->_position._x,
            nbody_cuda->_bodies[i]->_position._y,
            nbody_cuda->_bodies[i]->_position._z 
            );
            std::cout<<nbody_cuda->_bodies[i]->_position._x<<","<<
            nbody_cuda->_bodies[i]->_position._y<<","<<
            nbody_cuda->_bodies[i]->_position._z<<std::endl; 
    }
    std::cout<<std::endl;
}


// Render context, things that are just in the tuto
void setupRenderContext() 
{
   sShaderManager.InitializeStockShaders();
   glEnable( GL_DEPTH_TEST );
   setupBodies();
   glClearColor( sBackgroundColor[0],
                 sBackgroundColor[1],
                 sBackgroundColor[2],
                 sBackgroundColor[3] );
   glEnable( GL_LINE_SMOOTH );
}



// Main loop (thanks to the eric dave tutorial)
int main( int argc, char **argv ) 
{
    //Do the OpenGL stuff

    printf("Set up the window \n");
    setupWindow( argc, argv );

    printf("Reading callbacks (none for now) \n");
    if(!paused)
    {
      registerCallbacks();
    }
    // Initialize GLEW
    GLenum anError = glewInit();

    if( anError != 0 ) {
        fprintf( stderr, "GLEW Error: %s\n",
                glewGetErrorString( anError ) );

        if( DEBUG == true ) {
            cerr << " done" << endl;
        }

        return 1;
    }


    printf("The GL context \n");
    setupRenderContext();

    printf("The main loop \n");
    glutMainLoop();
    return 0;
}


/* int main() 
{
  NBody_cuda nbody = NBody_cuda(Nu,1.,2);

  std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
  
  
  
  std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

  //Computing the time
  std::cout << "Time taken by the CUDA kernel is " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;
} */