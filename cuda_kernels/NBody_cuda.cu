#include "hip/hip_runtime.h"
#include "Physics.cuh"

#include <vector>
#include <chrono>

//https://gist.github.com/lebedov/bca3c70e664f54cdf8c3cd0c28c11a0f

class NBody_cuda 
{
    private:
        int _N;
        float _tf;
        int _timeSteps;
        float _deltaT;
    public:
        std::vector<Body*> _bodies;
        
        NBody_cuda(int, float, int);
        ~NBody_cuda() 
        { 
            for(int i=0; _bodies.size(); i++)
            {
                delete _bodies[i];
            }
        };

        void print_cycle(float);
        void print_sim();

        //Physics
        void display_bodies();
        void setUP_cuda();

        //getters and setters
        int get_N() const {return _N;}
        int getTimeSteps() const {return _timeSteps;}; 
};


// I did not know which extension to use .cpp or .cu ??
// déclarons les dans le meme fichier


//constructor
NBody_cuda::NBody_cuda(int Nu , float tf, int timeSteps): _N(Nu), _tf(tf), _timeSteps(timeSteps) 
{
  std::cout<<"=====Constructed the NBody class in cuda===="<<std::endl;
};

// useless tbh
void NBody_cuda::display_bodies()
{
  for( int i = 0; i < _N; i++ ) 
  {
    printf("\nBody %d:\nMass: %f\nPosition(x ,y, z): %f, %f, %f\nVelocity(x, y, z): %f, %f, %f\nAcceleration(x ,y, z): %f, %f, %f\n\n",
      i + 1, 
      _bodies[i]->_mass,
      _bodies[i]->_position._x, _bodies[i]->_position._y, _bodies[i]->_position._z,

      _bodies[i]->_velocity._x, _bodies[i]->_velocity._y, _bodies[i]->_velocity._z,
      _bodies[i]->_acceleration._x, _bodies[i]->_acceleration._y, _bodies[i]->_acceleration._z);
  }
};


// compute acceleration per thread
__device__
void d_updateAcceleration(int index, vector *d_pos,vector *d_acc, Mass *d_mass, int _N) 
{
   
  vector netForce = { 0, 0, 0 };

  for( int i = 0; i < _N; i++ ) 
  {
    if( i == index ) 
    {
      continue;
    }

    vector vectorForceToOther = {0, 0, 0};

    Force scalarForceBetween = ComputeForce(
                                  d_mass[index],
                                  d_mass[i],
                                  d_pos[index],
                                  d_pos[i]);



    v_direction( d_pos[index],d_pos[i],vectorForceToOther);

    vectorForceToOther._x *= scalarForceBetween;
    vectorForceToOther._y *= scalarForceBetween;
    vectorForceToOther._z *= scalarForceBetween;

    netForce._x += vectorForceToOther._x;
    netForce._y += vectorForceToOther._y;
    netForce._z += vectorForceToOther._z;

    printf("The force %d: %f %f %f \n", index, netForce._x, netForce._y, netForce._z);
  }

  d_acc[index] = computeAcceleration3D(d_mass[index], netForce);

  printf("The accelerations: %d : %f %f %f \n", index,d_acc[index]._x, d_acc[index]._y, d_acc[index]._z);
};

__device__
void d_updateVelocity(int index, float deltaT, vector *d_acc, vector *d_vel) 
{
  d_vel[index] = computeVelocity3D(
                                d_acc[index],
                                d_vel[index],
                                deltaT);
  printf("The velocities: %d : %f %f %f \n", index,d_vel[index]._x, d_vel[index]._y, d_vel[index]._z);
};

__device__
void d_updatePosition(int index, float deltaT, vector *d_vel, vector *d_pos) 
{

  d_pos[index] = computePosition3D( 
                              d_vel[index],
                              d_pos[index],
                              deltaT);
  printf("The positions: %d : %f %f %f \n", index,d_pos[index]._x, d_pos[index]._y, d_pos[index]._z);
};


__global__
void updatePhysics(int bodies, float deltaT, vector *d_pos, vector *d_vel, vector *d_acc, Mass *d_mass, int _N)
{
  
  // 1D blocks and thread organisation
  int blockidx = blockIdx.x;
  int threadidx = threadIdx.x;

  int element_id = (blockidx * threadidx) + threadidx;

  printf("the element id %d \n", element_id);

  if(element_id > _N)
    return;

  //printf("%s \n", "===== The positions in the updatePhys b4==== \n");
  for(int i=1; i<element_id; i++)
  {
    d_pos++;
  }
  printf("@ %d %f, %f, %f \n",element_id,d_pos->_x,d_pos->_y,d_pos->_z);


  d_updateAcceleration(element_id, d_pos, d_acc, d_mass, _N);
  d_updateVelocity(element_id, deltaT, d_acc, d_vel);
  d_updatePosition(element_id, deltaT, d_vel, d_pos);

  //printf("%s \n", "===== The positions in the updatePhys==== \n");
  for(int i=0; i<element_id; i++)
  {
    d_pos++;
  }
  printf("* %d %f, %f, %f \n",element_id,d_pos->_x,d_pos->_y,d_pos->_z);
};


// The execution is very different 
void NBody_cuda::setUP_cuda()
{  

  // the C syntax
  int VECTOR_SIZE_IN_BYTES = _N * sizeof(vector);
  int SCALAR_SIZE_IN_BYTES = _N * sizeof(Scalar);

  //Initializing Velocities of N bodies in GPU
  vector *h_vel = nBodyVelocity;
  vector *d_vel;
  hipMalloc((void**) &d_vel, VECTOR_SIZE_IN_BYTES);
  hipMemcpy(d_vel, h_vel, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice);

  //Initializing acceleration of N bodies in GPU
  vector *h_acc = nBodyAcceleration;
  vector *d_acc;
  hipMalloc((void**) &d_acc, VECTOR_SIZE_IN_BYTES);
  hipMemcpy(d_acc, h_acc, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice);
  
  //Initializing Mass of N bodies in GPU
  Mass *h_mass = nBodyMass;
  Mass *d_mass;
  hipMalloc((void**) &d_mass, SCALAR_SIZE_IN_BYTES);
  hipMemcpy(d_mass, h_mass, SCALAR_SIZE_IN_BYTES, hipMemcpyHostToDevice);

    //Initializing Positions of N bodies in GPU
  vector *h_pos = nBodyPosition;
  vector *d_pos;
  hipMalloc((void**) &d_pos, VECTOR_SIZE_IN_BYTES);
  hipMemcpy(d_pos, h_pos, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice);


  std::cout<<"===== The starting positions===="<<std::endl;
  for(int i=0; i<_N; i++)
  {
    std::cout<<h_pos->_x<<","<<h_pos->_y<<","<<h_pos->_z<<std::endl;
    h_pos++;
  }



  for (int i = 0; i < 1; ++i)
  {
    printf("The number of 32 thread blocks %d \n", (int)ceil(_N/32));

    std::cout << "AT time step "<<(float)(100 * i) << std::endl;
    updatePhysics<<<(int)ceil(_N/32), 32>>>(_N, (float)(i * 100), d_pos, d_vel, d_acc, d_mass, _N);
  }
    

  hipMemcpy(h_pos, d_pos, VECTOR_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_vel, d_vel, VECTOR_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_acc, d_acc, VECTOR_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
  hipFree(d_pos);
  hipFree(d_vel);
  hipFree(d_acc);
  hipFree(d_mass);

  std::cout<<"===== The Final positions===="<<std::endl;
  for(int i=0; i<_N; i++)
  {
    std::cout<<h_pos->_x<<","<<h_pos->_y<<","<<h_pos->_z<<std::endl;
    h_pos++;
  }

  

}

int main() 
{
  NBody_cuda nbody = NBody_cuda(N,1.,2);

  std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
  
  nbody.setUP_cuda();
  
  std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

  //Computing the time
  std::cout << "Time taken by the CUDA kernel is " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;
}