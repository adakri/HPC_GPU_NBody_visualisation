#include "hip/hip_runtime.h"
#include "Physics.cuh"

#include <vector>

//https://gist.github.com/lebedov/bca3c70e664f54cdf8c3cd0c28c11a0f

class NBody_cuda 
{
    private:
        int _N;
        float _tf;
        int _timeSteps;
        float _deltaT;
    public:
        std::vector<Body*> _bodies;
        
        NBody_cuda(int, float, int);
        ~NBody_cuda() 
        { 
            for(int i=0; _bodies.size(); i++)
            {
                delete _bodies[i];
            }
        }

        void print_cycle(float);
        void print_sim();

        //Physics
        void display_bodies();
        void setUP_cuda();

        //getters and setters
        int get_N() const {return _N;}
        int getTimeSteps() const {return _timeSteps;}; 
};


// I did not know which extension to use .cpp or .cu ??
// déclarons les dans le meme fichier


//constructor


void NBody_cuda::display_bodies()
{
  for( int i = 0; i < _N; i++ ) 
  {
    printf("\nBody %d:\nMass: %f\nPosition(x ,y, z): %f, %f, %f\nVelocity(x, y, z): %f, %f, %f\nAcceleration(x ,y, z): %f, %f, %f\n\n",
      i + 1, 
      _bodies[i]->_mass,
      _bodies[i]->_position._x, _bodies[i]->_position._y, _bodies[i]->_position._z,

      _bodies[i]->_velocity._x, _bodies[i]->_velocity._y, _bodies[i]->_velocity._z,
      _bodies[i]->_acceleration._x, _bodies[i]->_acceleration._y, _bodies[i]->_acceleration._z);
  }
};


// compute acceleration per thread
__device__
void d_updateAcceleration(int index, Position3D *d_pos,Acceleration3D *d_acc, Mass *d_mass, int _N) 
{
   
  vector netForce = { 0, 0, 0 };

  for( int i = 0; i < _N; i++ ) 
  {
    if( i == index ) 
    {
      continue;
    }

    vector vectorForceToOther = {0, 0, 0};

    Force scalarForceBetween = forceNewtonianGravity3D(
                                  d_mass[index],
                                  d_mass[i],
                                  d_pos[index],
                                  d_pos[i]);

    direction( d_pos[index],d_pos[i],vectorForceToOther);

    vectorForceToOther._x *= scalarForceBetween;
    vectorForceToOther._y *= scalarForceBetween;
    vectorForceToOther._z *= scalarForceBetween;

    netForce._x += vectorForceToOther._x;
    netForce._y += vectorForceToOther._y;
    netForce._z += vectorForceToOther._z;
  }

  d_acc[index] = computeAccel3D(d_mass[index], netForce);
};

__device__
void d_updateVelocity(int index, float deltaT, Acceleration3D *d_acc, Velocity3D *d_vel) 
{
  d_vel[index] = computeVelo3D(
                                d_acc[index],
                                d_vel[index],
                                deltaT);
};

__device__
void d_updatePosition(int index, float deltaT, Velocity3D *d_vel, Position3D *d_pos) 
{

  d_pos[index] = computePos3D( 
                              d_vel[index],
                              d_pos[index],
                              deltaT);
};


__global__
void updatePhysics(int bodies, float deltaT, Position3D *d_pos, Velocity3D *d_vel, Acceleration3D *d_acc, Mass *d_mass, int _N)
{
  
  // 1D blocks and thread organisation
  int blockidx = blockIdx.x;
  int threadidx = threadIdx.x;

  int element_id = (blockidx * threadidx) + threadidx;

  if(element_id > _N)
    return;
  
  d_updateAcceleration(element_id, d_pos, d_acc, d_mass, _N);
  d_updateVelocity(element_id, deltaT, d_acc, d_vel);
  d_updatePosition(element_id, deltaT, d_vel, d_pos);
};


// The execution is very different 
void NBody_cuda::setUP_cuda()
{
  double start, end, min = 1e5;

  // the C syntax
  int VECTOR_SIZE_IN_BYTES = _N * sizeof(vector);
  int SCALAR_SIZE_IN_BYTES = _N * sizeof(Scalar);

  //Initializing Velocities of N bodies in GPU
  Velocity3D *h_vel = nBodyVelocity;
  Velocity3D *d_vel;
  hipMalloc((void**) &d_vel, VECTOR_SIZE_IN_BYTES);
  hipMemcpy(d_vel, h_vel, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice);

  //Initializing acceleration of N bodies in GPU
  Acceleration3D *h_acc = nBodyAcceleration;
  Acceleration3D *d_acc;
  hipMalloc((void**) &d_acc, VECTOR_SIZE_IN_BYTES);
  hipMemcpy(d_acc, h_acc, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice);
  
  //Initializing Mass of N bodies in GPU
  Mass *h_mass = nBodyMass;
  Mass *d_mass;
  hipMalloc((void**) &d_mass, SCALAR_SIZE_IN_BYTES);
  hipMemcpy(d_mass, h_mass, SCALAR_SIZE_IN_BYTES, hipMemcpyHostToDevice);

    //Initializing Positions of N bodies in GPU
  Position3D *h_pos = nBodyPosition;
  Position3D *d_pos;
  hipMalloc((void**) &d_pos, VECTOR_SIZE_IN_BYTES);
  hipMemcpy(d_pos, h_pos, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice);


  for (int j = 0; j < 3; ++j)
  {
    for (int i = 0; i < 10000; ++i)
    {
      updatePhysics<<<(_N/16) + 1, 16>>>(_N, (float)(i * 100), d_pos, d_vel, d_acc, d_mass, _N);
    }
    
  }
  hipMemcpy(h_pos, d_pos, VECTOR_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_vel, d_vel, VECTOR_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(h_acc, d_acc, VECTOR_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
  hipFree(d_pos);
  hipFree(d_vel);
  hipFree(d_acc);
  hipFree(d_mass);

  printf("Time Taken by CUDA implementation: %f ms\n", (min)*1000);
}